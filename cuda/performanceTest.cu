#include "hip/hip_runtime.h"

/****   AUTHOR INFORMATION
 
 NAME : Alex Valerio Andriati
 AFFILIATION : University of Sao Paulo - Brazil

 Last update : November/02/2019

 -------------------------------------------------------------------------

 ****  TEST ROUTINES TO COMPUTE PHYSICAL OPERATORS
 *
 * COMPILE :
 *
 * nvcc performanceTest.c -o exe
 *
 * HOW TO EXECUTE :
 *
 * ./exe Nparticles Norbitals
 *
 * where 'Nparticles' and 'Morbitals' are command line arguments for the
 * number of particles and individual particle states respectively.
 *
 * In order to measure time one may use the command nvproof before ./exe
 * For more information check out CUDA developer guide
 *
 * ----------------------------------------------------------------------- */

#include "hamiltonianMatrix.cuh"



int main(int argc, char * argv[])
{

    int
        i,
        j,
        q,
        l,
        nc,
        Npar,
        Morb,
        blocks;

    size_t
        nbytes;

    double
        sum,
        realPart,
        imagPart;

    hipError_t
        err;

    hipDoubleComplex
        z;

    Iarray
        Map,
        d_Map,
        MapOT,
        d_MapOT,
        MapTT,
        d_MapTT,
        IFmat,
        d_IFmat,
        NCmat,
        strideOT,
        d_strideOT,
        strideTT,
        d_strideTT;

    Carray
        C,
        d_C,
        out,
        d_out,
        Ho,
        d_Ho,
        Hint,
        d_Hint;



    if (argc != 3)
    {
        printf("\n\nERROR: Need two integer numbers from command line ");
        printf("the first number of particles and second the number of ");
        printf("orbitals.\n\n");
        exit(EXIT_FAILURE);
    }

    err = hipSuccess;

    sscanf(argv[1],"%d",&Npar);
    sscanf(argv[2],"%d",&Morb);
    nc = NC(Npar,Morb);

    NCmat = setupNCmat(Npar,Morb);
    IFmat = setupFocks(Npar,Morb);

    // alloc NCmat and IFmat on device

    cuda_iarrDef( nc * Morb , &d_IFmat );

    strideTT = iarrDef(nc);
    strideOT = iarrDef(nc);

    // alloc Map's strides on device

    cuda_iarrDef( nc , &d_strideTT );
    cuda_iarrDef( nc , &d_strideOT );

    Map = OneOneMap(Npar,Morb,NCmat,IFmat);
    MapTT = TwoTwoMap(Npar,Morb,NCmat,IFmat,strideTT);
    MapOT = OneTwoMap(Npar,Morb,NCmat,IFmat,strideOT);

    // alloc Maps on device

    cuda_iarrDef( nc * Morb * Morb , &d_Map );
    cuda_iarrDef( strideOT[nc-1] + Morb*Morb , &d_MapOT );
    cuda_iarrDef( strideTT[nc-1] , &d_MapTT );

    // Copy data from host to device

    nbytes = nc * Morb * sizeof(int);
    err = hipMemcpy(d_IFmat,IFmat,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    nbytes = nc * sizeof(int);
    err = hipMemcpy(d_strideOT,strideOT,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    nbytes = nc * sizeof(int);
    err = hipMemcpy(d_strideTT,strideTT,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    nbytes = nc * Morb * Morb * sizeof(int);
    err = hipMemcpy(d_Map,Map,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    nbytes = (strideOT[nc-1] + Morb*Morb) * sizeof(int);
    err = hipMemcpy(d_MapOT,MapOT,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    nbytes = strideTT[nc-1] * sizeof(int);
    err = hipMemcpy(d_MapTT,MapTT,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("\nNumber of particles : %3d", Npar);
    printf("\nNumber of orbitals  : %3d", Morb);
    printf("\nNumber of configurations : %d", nc);

    printf("\n\n======================================\n\n");

    printf("MEMORY CONSUMPTION (in Mb)");

    printf("\n\nMemory for coefficients : %.1lf",
            ((double) 2*nc*sizeof(double)) / 1E6);

    printf("\nMemory for Fock states : %.1lf",
            ((double) nc*Morb*sizeof(int)) / 1E6);

    printf("\nMemory for single jump from 1 orbital Map : %.1lf",
            ((double) nc*Morb*Morb*sizeof(int)) / 1E6);

    printf("\nMemory for double jump from 1 orbital Map : %.1lf",
            ((double) strideOT[nc-1]*sizeof(int))/1E6);
    printf("\nMemory for double jump from 2 orbitals Map : %.1lf",
            ((double) strideTT[nc-1]*sizeof(int))/1E6);



    Ho = carrDef(Morb*Morb);

    Hint = carrDef(Morb*Morb*Morb*Morb);

    C = carrDef(nc);

    out = carrDef(nc);

    // alloc device arrays
    cuda_carrDef(nc,&d_out);
    cuda_carrDef(nc,&d_C);
    cuda_carrDef(Morb*Morb,&d_Ho);
    cuda_carrDef(Morb*Morb*Morb*Morb,&d_Hint);

    sum = 0.0;
    for (i = 0; i < nc; i++)
    {
        realPart = sin( 20 * ((double) i) / nc) * (i % 13);
        imagPart = (i % 8) - (i % 3);
        C[i] = make_hipDoubleComplex(realPart,imagPart);
        sum = sum + realPart * realPart + imagPart * imagPart;
    }

    // normalize to 1
    for (i = 0; i < nc; i++)
    {
        C[i] = hipCdiv(C[i],make_hipDoubleComplex(sqrt(sum),0));
    }

    // Done in the Host copy to device
    nbytes = nc * sizeof(hipDoubleComplex);
    err = hipMemcpy(d_C,C,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    for (i = 0; i < Morb; i++)
    {
        Ho[i + Morb*i] = make_hipDoubleComplex((i % 4) - 1,0);
        for (j = i + 1; j < Morb; j++)
        {
            realPart = i * (j % 3) - (i % 4) + 5 * (j % 2);
            imagPart = -4.123 * i / (j + 1);
            Ho[i + Morb*j] = make_hipDoubleComplex(realPart,imagPart);
            Ho[j + Morb*i] = hipConj(Ho[i + Morb*j]);
        }
    }

    for (i = 0; i < Morb*Morb*Morb*Morb; i++)
    {
        Hint[i] = make_hipDoubleComplex(1.234,0);
    }

    for (i = 0; i < Morb; i++)
    {
        for (j = i + 1; j < Morb; j++)
        {
            for (q = 0; q < Morb; q++)
            {
                if (q == i || q == j) continue;
                for (l = q + 1; l < Morb; l++)
                {
                    if (l == i || l == j) continue;
                    // real part
                    realPart = i - 2 + 10 * (j % (i+1)) - q * l;
                    // imag part
                    imagPart = ((double) i * q - j * l) / Morb;

                    z = make_hipDoubleComplex(realPart,imagPart);
                    Hint[i+j*Morb+q*Morb*Morb+l*Morb*Morb*Morb] = z;
                    Hint[i+j*Morb+l*Morb*Morb+q*Morb*Morb*Morb] = z;
                    Hint[j+i*Morb+l*Morb*Morb+q*Morb*Morb*Morb] = z;
                    Hint[j+i*Morb+q*Morb*Morb+l*Morb*Morb*Morb] = z;
                    Hint[q+l*Morb+i*Morb*Morb+j*Morb*Morb*Morb] = hipConj(z);
                    Hint[q+l*Morb+j*Morb*Morb+i*Morb*Morb*Morb] = hipConj(z);
                    Hint[l+q*Morb+i*Morb*Morb+j*Morb*Morb*Morb] = hipConj(z);
                    Hint[l+q*Morb+j*Morb*Morb+i*Morb*Morb*Morb] = hipConj(z);
                }
            }
        }
    }

    // Copy values of matrix elements to device

    nbytes = Morb * Morb * sizeof(hipDoubleComplex);
    err = hipMemcpy(d_Ho,Ho,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    nbytes = Morb * Morb * Morb * Morb * sizeof(hipDoubleComplex);
    err = hipMemcpy(d_Hint,Hint,nbytes,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Host to Device - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    printf("\n\n======================================\n\n");

    // optimally choose the number of blocks
    blocks = (nc + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    for (i = 0; i < 5; i++)
    {
        applyHconf<<<blocks,THREADS_PER_BLOCK>>>(Npar,Morb,d_Map,d_MapOT,d_MapTT,
            d_strideOT,d_strideTT,d_IFmat,d_C,d_Ho,d_Hint,d_out);

        hipDeviceSynchronize();
    }

    nbytes = nc * sizeof(hipDoubleComplex);
    err = hipMemcpy(out,d_out,nbytes,hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("\n\nFailed to copy data from Device to Host - ");
        printf(" error code : %s!\n\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // free Host memory
    free(C);
    free(out);
    free(Ho);
    free(Hint);
    free(IFmat);
    free(NCmat);
    free(strideOT);
    free(strideTT);
    free(MapOT);
    free(MapTT);
    free(Map);

    // free device memory

    hipFree(d_C);
    hipFree(d_out);
    hipFree(d_Ho);
    hipFree(d_Hint);
    hipFree(d_IFmat);
    hipFree(d_strideOT);
    hipFree(d_strideTT);
    hipFree(d_MapOT);
    hipFree(d_MapTT);
    hipFree(d_Map);

    printf("\n\nDone.\n\n");
    return 0;
}
